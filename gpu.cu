#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <math.h>


using namespace std;
using namespace chrono;


__device__ double f(double x) {
    return pow(x, 4) - 3 * pow(x, 3) + 2 * pow(x, 2) + x - 5;
}


__global__ void simpson(double* x, int n, double lower_limit, double upper_limit) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        double x0 = lower_limit + i * (upper_limit - lower_limit) / n;
        double x1 = x0 + (upper_limit - lower_limit) / n;
        double h = (x1 - x0) / 2.0;
        double y0 = f(x0);
        double y1 = f(x1);
        if (i == 0 || i == n-1) {
            x[i] = y0 + y1;
        } else if (i % 2 == 0) {
            x[i] = 4.0 * y0 + 2.0 * y1;
        } else {
            x[i] = 2.0 * y0 + y1;
        }
    }
}

int main() {
    int n = 1000000; 
    double lower_limit = 0.0;
    double upper_limit = 10.0; 
    double h = (upper_limit - lower_limit) / n;
    double* x;
    double result = 0.0;
    

    x = new double[n];

    double* d_x;
    hipMalloc((void**)&d_x, sizeof(double) * n);
    auto start = std::chrono::high_resolution_clock::now();
    
    simpson<<<(n + 255) / 256, 256>>>(d_x, n, lower_limit, upper_limit);

    hipMemcpy(x, d_x, sizeof(double) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        result += x[i];
    }
    result *= h / 4.5;
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::nano> elapsed = end - start;
    std::cout << "Elapsed time: " << elapsed.count() * 1e-6 << " milliseconds" << std::endl;
    
    std::cout << "The area under the curve from " << lower_limit << " to " << upper_limit << " is: " << result << std::endl;


    delete[] x;
    hipFree(d_x);

    return 0;
}

